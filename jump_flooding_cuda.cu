#include "hip/hip_runtime.h"
#include <limits>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_X 16
#define BLOCK_Y 16


__global__ void initKernel(float* input, int3* output, int2 size) {
    int x = int(blockIdx.x * blockDim.x + threadIdx.x);
	int y = int(blockIdx.y * blockDim.y + threadIdx.y);
    if (x >= size.x || y >= size.y) return;

    int pixelIdx = x * size.y + y;
    float inputValue = input[pixelIdx];
    if (inputValue < 0.0f) output[pixelIdx] = make_int3(x, y, 1); // interior
    else output[pixelIdx] = make_int3(-1, -1, -1);
}


__global__ void jumpFloodingKernel(int3* input, int3* output, int step, int2 size) {
	int x = int(blockIdx.x * blockDim.x + threadIdx.x);
	int y = int(blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= size.x || y >= size.y) return;

    int selfIdx = x * size.y + y;
    int3 inputData = input[selfIdx];
    if (inputData.z == 1) { // interior
        output[selfIdx] = inputData;
        return;
    }

    int2 nearestCoord = make_int2(inputData.x, inputData.y);
    float minSquareDist = std::numeric_limits<float>::max();
    if (nearestCoord.x != -1 && nearestCoord.y != -1) {
        float offsetX = float(nearestCoord.x - x);
        float offsetY = float(nearestCoord.y - y);
        minSquareDist = offsetX * offsetX + offsetY * offsetY;
    }

    for (int gridY = -1; gridY < 2; ++gridY) {
        for (int gridX = -1; gridX < 2; ++gridX) {
            if (gridX == 0 && gridY == 0) continue; // skip self

            int xLookup = x + gridX * step;
            int yLookup = y + gridY * step;

            if (xLookup < 0 || xLookup >= size.x || yLookup < 0 || yLookup >= size.y) continue; // out of bounds

            int lookupIdx = xLookup * size.y + yLookup;
            inputData = input[lookupIdx];
            int2 coord = make_int2(inputData.x, inputData.y);

            if (coord.x != -1 && coord.y != -1) {
                float offsetX = float(coord.x - x);
                float offsetY = float(coord.y - y);
                float squareDist = offsetX * offsetX + offsetY * offsetY;
                if (squareDist < minSquareDist) {
                    nearestCoord = coord;
                    minSquareDist = squareDist;
                }
            }
        }
    }

    output[selfIdx] = make_int3(nearestCoord.x, nearestCoord.y, -1);
}


__global__ void signedDistanceKernel(int3* input, float2* output, int2 size) {
    int x = int(blockIdx.x * blockDim.x + threadIdx.x);
	int y = int(blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= size.x || y >= size.y) return;

    int idx = x * size.y + y;
    int3 inputData = input[idx];
    if (inputData.z == 1) output[idx] = make_float2(0.0f, 0.0f);
    else {
        output[idx] =  make_float2(
            float(inputData.x - x),
            float(inputData.y - y)
        );
    }
}


void initDataCuda(float* input, int* output, int H, int W) {
    dim3 grid((H + BLOCK_X - 1) / BLOCK_X, (W + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);
    int2 size = make_int2(H, W);
    initKernel <<< grid, block >>> (input, (int3*)output, size);
}


bool jumpFloodingCuda(int* buffer1, int* buffer2, int H, int W) {
    dim3 grid((H + BLOCK_X - 1) / BLOCK_X, (W + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

    bool reverse = true;
    int step = max(H, W);
    int2 size = make_int2(H, W);

    while (step > 1) {
        reverse = !reverse;
        step = (step + 1) >> 1;

        if (reverse) jumpFloodingKernel <<< grid, block >>> ((int3*)buffer2, (int3*)buffer1, step, size);
        else jumpFloodingKernel <<< grid, block >>> ((int3*)buffer1, (int3*)buffer2, step, size);
    }
    return reverse;
}


void signedDistanceCuda(int* buffer, float* output, int H, int W) {
    dim3 grid((H + BLOCK_X - 1) / BLOCK_X, (W + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);
    int2 size = make_int2(H, W);
    signedDistanceKernel <<< grid, block >>> ((int3*)buffer, (float2*)output, size);
}